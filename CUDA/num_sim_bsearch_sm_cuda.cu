#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <sys/time.h>


#include "../lib/bed.h"
#include "../lib/set_intersect.h"
#include "radixsort.h"
//#include "gpu.hpp"
#include "random.hpp"
#include "../lib/timer.h"

#include "set_intersect_cuda.h"

int main(int argc, char *argv[]) {

	hipFree(NULL);

	if (argc < 6) {
		fprintf(stderr, "usage: order <u> <a> <b> <reps> <inter N> <sum N>\n"
						"e.g., order U.bed A.bed B.bed 10000 1 1024\n");
		return 1;
	}

	int chrom_num = 24;

	/***********************REPLACE WITH INPUT FILE************************/	
	char *chrom_names[] = {
		"chr1",  "chr2",  "chr3", "chr4",  "chr5",  "chr6",  "chr7", "chr8",
		"chr9", "chr10", "chr11", "chr12", "chr13", "chr14", "chr15", "chr16",
		"chr17", "chr18", "chr19", "chr20", "chr21", "chr22", "chrX",  "chrY"
	};
	/**********************************************************************/	

	struct chr_list *U, *A, *B;

	char *U_file = argv[1], *A_file = argv[2], *B_file = argv[3];
	int reps = atoi(argv[4]);
	int inter_threads = atoi(argv[5]);
	int sum_threads = atoi(argv[6]);

	if	( ( chr_list_from_bed_file(&U, chrom_names, chrom_num, U_file) == 1) ||
		  ( chr_list_from_bed_file(&A, chrom_names, chrom_num, A_file) == 1) ||
		  ( chr_list_from_bed_file(&B, chrom_names, chrom_num, B_file) == 1) ) {
		fprintf(stderr, "Error parsing bed files.\n");
		return 1;
	}

	unsigned int max = add_offsets(U, chrom_num);

	trim(U, A, chrom_num);
	trim(U, B, chrom_num);

	int A_size, B_size, U_size;

	struct bed_line *U_array, *A_array, *B_array;

	U_size = chr_array_from_list(U, &U_array, chrom_num);
	A_size = chr_array_from_list(A, &A_array, chrom_num);
	B_size = chr_array_from_list(B, &B_array, chrom_num);

	unsigned int *A_key_h = 
		(unsigned int *) malloc( (A_size) * sizeof(unsigned int));
	unsigned int *A_val_h = 
		(unsigned int *) malloc( (A_size) * sizeof(unsigned int));

	unsigned int *B_key_h = 
		(unsigned int *) malloc( (B_size) * sizeof(unsigned int));
	unsigned int *B_val_h = 
		(unsigned int *) malloc( (B_size) * sizeof(unsigned int));


	/*
	 * In CUDA we can sort key value pairs, 
	 * the key can be the offset, and the value can be the length
	 */
	set_start_len( U_array, U_size,
				   A_array, A_key_h, A_val_h, A_size );

	set_start_len( U_array, U_size,
				   B_array, B_key_h, B_val_h, B_size );

	// Move A and B to deviceB
	unsigned int *A_key_d, *A_val_d, *B_key_d, *B_val_d;
	hipMalloc((void **)&A_key_d, (A_size)*sizeof(unsigned int));
	hipMalloc((void **)&A_val_d, (A_size)*sizeof(unsigned int));
	hipMalloc((void **)&B_key_d, (B_size)*sizeof(unsigned int));
	hipMalloc((void **)&B_val_d, (B_size)*sizeof(unsigned int));

	start();
	hipMemcpy(A_key_d, A_key_h, (A_size) * sizeof(unsigned int), 
			hipMemcpyHostToDevice);
	hipMemcpy(A_val_d, A_val_h, (A_size) * sizeof(unsigned int),
			hipMemcpyHostToDevice);
	hipMemcpy(B_key_d, B_key_h, (B_size) * sizeof(unsigned int), 
			hipMemcpyHostToDevice);
	hipMemcpy(B_val_d, B_val_h, (B_size) * sizeof(unsigned int),
			hipMemcpyHostToDevice);
	stop();

	unsigned long memup_time = report();

	int block_size = 256;
	dim3 dimBlock(block_size);

	// R will hold the results of the intersection, for each interval A[i],
	// R[i] will be the number of intervals in B that A[i] intersects,
	unsigned int *R_d;
	hipMalloc((void **)&R_d, (A_size)*sizeof(unsigned int));

	// *_key_d holds the start position, and *_val_d holds the length,
	// the end position is *_key_d + *_val_d
	//
	// Each thread will search |reps| items in A, we will keep the blocksize
	// fixed at 256, but we will need to adjust the grid size 
	
	int grid_size = ( A_size + block_size - 1) / (block_size * 1);
	dim3 dimGridSearch( grid_size );

	hipError_t err;

	// Sort A
	nvRadixSort::RadixSort radixsortA(A_size, false);
	radixsortA.sort((unsigned int*)A_key_d, (unsigned int*)A_val_d, 
			A_size, 32);

	// Sort B
	nvRadixSort::RadixSort radixsortB(B_size, false);
	radixsortB.sort((unsigned int*)B_key_d, (unsigned int*)B_val_d, 
			B_size, 32);
	hipDeviceSynchronize();
	stop();

	unsigned int *R_h = (unsigned int *) malloc( A_size * sizeof(unsigned int));

	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort: %s.\n", hipGetErrorString( err) );

	start();
	intersection_b_search_sm <<< dimGridSearch, 
								 dimBlock,
								 2000 * sizeof(unsigned int)
								>>> ( A_key_d, A_val_d, A_size,
										   B_key_d, B_val_d, B_size,
										   R_d, 1);

	hipDeviceSynchronize();
	parallel_sum(R_d, block_size, A_size, sum_threads);
	stop();


	unsigned int O;
	start();
	hipMemcpy(&O, R_d, 1 * sizeof(unsigned int), hipMemcpyDeviceToHost);
	stop();
	unsigned long memdown_time = report();


	hipFree(A_key_d);
	hipFree(B_key_d);

	srand(time(NULL));	

	RNG_rand48 A_r(rand());
	RNG_rand48 B_r(rand());
	dim3 dimGridAR( ceil(float(A_size)/float(dimBlock.x)));
	dim3 dimGridBR( ceil(float(B_size)/float(dimBlock.x)));
	nvRadixSort::RadixSort radixsortAR(A_size, true);
	nvRadixSort::RadixSort radixsortBR(B_size, true);

	int i,r = 0;
	unsigned int R;
	unsigned int *A_r_d, *B_r_d;

	unsigned long rand_total_time = 0,
				  sort_total_time = 0,
				  intersect_total_time = 0;

	for  (i = 0; i < reps; i ++) {
		start();
		A_r.generate(A_size);

		hipDeviceSynchronize();
		err = hipGetLastError();
		if(err != hipSuccess)
			fprintf(stderr, "Rand A: %s.\n", hipGetErrorString( err) );


		B_r.generate(B_size);

		hipDeviceSynchronize();
		err = hipGetLastError();
		if(err != hipSuccess)
			fprintf(stderr, "Rand B: %s.\n", hipGetErrorString( err) );

		A_r_d = (unsigned int *)A_r.get_random_numbers();
		B_r_d = (unsigned int *)B_r.get_random_numbers();

		normalize_rand <<<dimGridAR, dimBlock>>> (A_r_d, max, A_size);

		hipDeviceSynchronize();
		err = hipGetLastError();
		if(err != hipSuccess)
			fprintf(stderr, "Norm A: %s.\n", hipGetErrorString( err) );

		normalize_rand <<<dimGridBR, dimBlock>>> (B_r_d, max, B_size);

		hipDeviceSynchronize();
		err = hipGetLastError();
		if(err != hipSuccess)
			fprintf(stderr, "Norm B: %s.\n", hipGetErrorString( err) );
		stop();
		//printf("r:%ld\t", report());
		rand_total_time += report();

		start();
		radixsortAR.sort((unsigned int*)A_r_d, 0, A_size, 32);

		hipDeviceSynchronize();
		err = hipGetLastError();
		if(err != hipSuccess)
			fprintf(stderr, "Sort A: %s.\n", hipGetErrorString( err) );

		radixsortBR.sort((unsigned int*)B_r_d, 0, B_size, 32);

		hipDeviceSynchronize();
		err = hipGetLastError();
		if(err != hipSuccess)
			fprintf(stderr, "Sort B: %s.\n", hipGetErrorString( err) );
		stop();
		//printf("s:%ld\t", report());
		sort_total_time += report();

		intersection_b_search <<<dimGridSearch, 
							dimBlock >>> ( A_r_d, A_val_d, A_size,
										   B_r_d, B_val_d, B_size,
										   R_d, inter_threads);

		hipDeviceSynchronize();
		err = hipGetLastError();
		if(err != hipSuccess)
			fprintf(stderr, "intersect search: %s.\n", 
					hipGetErrorString(err) );

		start();
		parallel_sum( R_d, block_size, A_size, sum_threads);

		hipMemcpy(&R, R_d, 1 * sizeof(unsigned int), hipMemcpyDeviceToHost);
		stop();
		//printf("i:%ld\t", report());
		intersect_total_time += report();

		//printf("%ld\n", R);
		
		if (R >= O)
			++r;
	}


	double p = ( (double)(r + 1) ) / ( (double)(reps + 1) );
	fprintf(stderr,"O:%d\tp:%f\n", O, p);

	double  rand_avg_time = ( (double) rand_total_time) / reps,
			sort_avg_time = ( (double) sort_total_time) / reps,
			intersect_avg_time = ( (double)  intersect_total_time) / reps;

	double total_avg_time = rand_avg_time + sort_avg_time + intersect_avg_time;

	double  rand_prop_time = rand_avg_time/total_avg_time,
			sort_prop_time = sort_avg_time/total_avg_time,
			intersect_prop_time = intersect_avg_time/total_avg_time;

	printf("t:%G\tr:%G,%G\ts:%G,%G\ti:%G,%G\n", 
			total_avg_time,
			rand_avg_time, rand_prop_time,
			sort_avg_time, sort_prop_time,
			intersect_avg_time, intersect_prop_time);


	hipFree(A_key_d);
	hipFree(B_key_d);
	hipFree(R_d);

	return 0;
}
